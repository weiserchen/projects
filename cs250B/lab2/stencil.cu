
#include <hip/hip_runtime.h>
#include <time.h>
#include <stdio.h>
#include <iostream>
#include <math.h>

#include <ctime>
#include <chrono>

__device__
inline int idx(int x, int y, int width) {
	return y*width+x;
}

__global__ void stencil_kernel(float* temp, float* temp2, float* conduct, int w, int h) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int x = i % w, y = i / w;
	if (x > 0 && y > 0 && x < (w-1) && y < (h-1)) {
		float e = temp[i];
		temp2[i] = e +
			(
				(temp[i-1] - e) * conduct[i-1]
				+
				(temp[i+1] - e) * conduct[i+1]
				+
				(temp[i-w] - e) * conduct[i-w]
				+
				(temp[i+w] - e) * conduct[i+w]
			) * 0.2;
	}
}

void stencil_parallel(float* temp, float* temp2, float* conduct, int width, int height, int steps) {
	int n = width * height;
	// int threads = 1024;
	// int threads = 512;
	// int threads = 256;
	int threads = 128;
	// int threads = 64;
	int blocks = (n + threads - 1) / threads;

	for ( int s = 0; s < steps; s++ ) {
		stencil_kernel<<<blocks,threads>>>(temp, temp2, conduct, width, height);
		float* t = temp;
		temp = temp2;
		temp2 = t;
	}
}

__global__
void stencil_cuda(float* temp, float* temp2, float* conduct, int width, int height, int steps) {
	for ( int s = 0; s < steps; s++ ) {
		for ( int i = 1; i < width-1; i++ ) {
			for ( int j = 1; j < height-1; j++ ) {
				temp2[idx(i,j,width)] = 
					temp[idx(i,j,width)] +
					(
						(temp[idx(i-1,j,width)] - temp[idx(i,j,width)]) * conduct[idx(i-1,j,width)]
						+ 
						(temp[idx(i+1,j,width)] - temp[idx(i,j,width)]) * conduct[idx(i+1,j,width)]
						+ 
						(temp[idx(i,j-1,width)] - temp[idx(i,j,width)]) * conduct[idx(i,j-1,width)]
						+ 
						(temp[idx(i,j+1,width)] - temp[idx(i,j,width)]) * conduct[idx(i,j+1,width)]
					) * 0.2;
			}
		}
		float* t = temp;
		temp = temp2;
		temp2 = t;
	}
}

void stencil_optimized(float* temp, float* conduct, int width, int height, int steps) {
	int deviceCount;
	hipGetDeviceCount(&deviceCount);
	printf( "Number of CUDA devices: %d\n", deviceCount );
	for ( int i = 0; i < deviceCount; i++ ) {
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);
		printf( "%d: %s\n", i, prop.name );
		printf( "Max %d threads per block\n", prop.maxThreadsPerBlock );
	}


	printf( "Starting cuda for %d steps on %dx%dx\n", steps, width, height );
	//cudaMemcpy(da, ha, N*sizeof(float), cudaMemcpyHostToDevice);
	float* dtemp, *dtemp2, *dconduct;
	hipMalloc(&dtemp, width*height*sizeof(float));
	hipMalloc(&dtemp2, width*height*sizeof(float));
	hipMalloc(&dconduct, width*height*sizeof(float));

	hipMemcpy(dtemp, temp, width*height*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dconduct, conduct, width*height*sizeof(float), hipMemcpyHostToDevice);

	// stencil_cuda<<<1,1>>>(dtemp,dtemp2,dconduct,width,height,steps);
	stencil_parallel(dtemp,dtemp2,dconduct,width,height,steps);
	printf( "Started kernel\n" );
	hipDeviceSynchronize();
	
	if ( steps % 2 == 0 ) {
		hipMemcpy(temp, dtemp, width*height*sizeof(float), hipMemcpyDeviceToHost);
	} else {
		hipMemcpy(temp, dtemp2, width*height*sizeof(float), hipMemcpyDeviceToHost);
	}
}
